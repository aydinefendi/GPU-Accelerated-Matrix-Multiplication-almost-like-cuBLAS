#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err) { if (err != hipSuccess) { \
    printf("CUDA error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); exit(EXIT_FAILURE); }}

#define CUBLAS_CHECK(err) { if (err != HIPBLAS_STATUS_SUCCESS) { \
    printf("cuBLAS error at %s:%d\n", __FILE__, __LINE__); exit(EXIT_FAILURE); }}

int main() {
    int N = 1024;
    int M = 1024;
    int K = 1024;

    float *h_A = (float *)malloc(N * K * sizeof(float));
    float *h_B = (float *)malloc(K * M * sizeof(float));
    float *h_C = (float *)malloc(N * M * sizeof(float));

    for (int i = 0; i < N * K; ++i) h_A[i] = 1.0f;
    for (int i = 0; i < K * M; ++i) h_B[i] = 2.0f;

    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_A, N * K * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_B, K * M * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_C, N * M * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, N * K * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, K * M * sizeof(float), hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    float alpha = 1.0f, beta = 0.0f;

    // Create and start CUDA timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Matrix multiplication: C = A * B
    // A (N×K), B (K×M), C (N×M)
    CUBLAS_CHECK(hipblasSgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             M, N, K,
                             &alpha,
                             d_B, M,
                             d_A, K,
                             &beta,
                             d_C, M));

    // Stop CUDA timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_ms = 0.0f;
    hipEventElapsedTime(&elapsed_ms, start, stop);

    printf("cuBLAS sgemm time: %.3f ms\n", elapsed_ms);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_CHECK(hipMemcpy(h_C, d_C, N * M * sizeof(float), hipMemcpyDeviceToHost));

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}