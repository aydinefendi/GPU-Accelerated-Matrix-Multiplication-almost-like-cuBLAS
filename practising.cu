#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <assert.h>

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

#define TILE_WIDTH 4

__global__ void matrixMulKernel(float *A, float *B, float *C, int N);

int main(int argc, char *argv[])
{
    int N = 12;
    float *A = (float *)malloc(sizeof(float) * N *N);
    float *B = (float *)malloc(sizeof(float) * N *N);
    float *C = (float *)malloc(sizeof(float) * N *N);

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++) {
            A[i*N + j] = (rand() % 10) + 1;
            B[i*N + j] = (rand() % 10) + 1;
            C[i*N + j] = 0;
        }
    }

    float *d_A, *d_B, *d_C;
    hipError_t err = hipMalloc((void**)&d_A, sizeof(float) *N*N);
    CUDA_CHECK(err);
    err = hipMalloc((void**)&d_B, sizeof(float) *N*N);
    CUDA_CHECK(err);
    err = hipMalloc((void**)&d_C, sizeof(float) *N*N);
    CUDA_CHECK(err);

    err = hipMemcpy(d_A, A, sizeof(float) *N*N, hipMemcpyHostToDevice);
    CUDA_CHECK(err);
    err = hipMemcpy(d_B, B, sizeof(float) *N*N, hipMemcpyHostToDevice);
    CUDA_CHECK(err);

    dim3 block(4, 4, 1);
    dim3 grid(ceil(N/block.x), ceil(N/block.y), 1);
    
    matrixMulKernel<<<grid, block>>>(d_A, d_B, d_C, N);

    err = hipMemcpy(C, d_C, sizeof(float) *N*N, hipMemcpyDeviceToHost);
    CUDA_CHECK(err);

    printf("\nMatrix Multiplication Results:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%.2f * %.2f = %.2f\n", A[i*N + j], B[i*N + j], C[i*N + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);
    return 0;
}

__global__ void matrixMulKernel(float *A, float *B, float *C, int N)
{
    assert(TILE_WIDTH == blockDim.x);
    assert(TILE_WIDTH == blockDim.y);
    assert(N % TILE_WIDTH == 0);

    int by = blockIdx.y;
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int i = by * TILE_WIDTH + ty;
    int j = bx * TILE_WIDTH + tx;

    __shared__ float sh_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sh_B[TILE_WIDTH][TILE_WIDTH];

    float value = 0;

    for (int phase = 0; phase < (float) ceil(N/TILE_WIDTH);phase++)
    {
        if (i < N && (phase *TILE_WIDTH + tx) < N)
        {
            sh_A[ty][tx] = A[(i*N + (phase *TILE_WIDTH + tx))];
        }
        else 
        {
            sh_A[ty][tx] = 0.0f;
        }

        if (j < N && ((phase*TILE_WIDTH + ty)*N + j) < N)
        {
            sh_B[ty][tx] = B[((phase*TILE_WIDTH + ty)*N + j)];
        }
        else
        {
            sh_B[ty][tx] = 0.0f;
        }
        __synchthreads;

        for (int k = 0; k < TILE_WIDTH;k++)
        {
            value += sh_A[ty][k] * sh_B[k][tx];
        }
        __synchthreads;
    }

    if (i < N && j < N)
    {
        C[i*N+j] = value;
    }
}
