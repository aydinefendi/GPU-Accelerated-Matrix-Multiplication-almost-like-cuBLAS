#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <assert.h>

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

#define TILE_WIDTH 64 

__global__ void matrixMulKernel(float *A, float *B, float *C, int N1, int N2, int N3);

int main(int argc, char *argv[])
{
    int N1 = 4096;
    int N2 = 4096; 
    int N3 = 4096;
    float *A = (float *)malloc(sizeof(float) * N1 *N2);
    float *B = (float *)malloc(sizeof(float) * N2 *N3);
    float *C = (float *)malloc(sizeof(float) * N1 *N3);

    for (int i = 0; i < N1; i++)
    {
        for (int j = 0; j < N2; j++) {
            A[i*N2 + j] = (rand() % 10) + 1;
        }
    }

    for (int i = 0; i < N2; i++)
    {
        for (int j = 0; j < N3; j++)
        {
            B[i*N3 + j] = (rand() % 10) + 1;
        }
    }

    for (int i = 0; i < N1; i++)
    {
        for (int j = 0; j < N3; j++)
        {
            C[i*N3 + j] = 0;
        }
    }

    float *d_A, *d_B, *d_C;
    hipError_t err = hipMalloc((void**)&d_A, sizeof(float) *N1*N2);
    CUDA_CHECK(err);
    err = hipMalloc((void**)&d_B, sizeof(float) *N2*N3);
    CUDA_CHECK(err);
    err = hipMalloc((void**)&d_C, sizeof(float) *N1*N3);
    CUDA_CHECK(err);

    err = hipMemcpy(d_A, A, sizeof(float) *N1*N2, hipMemcpyHostToDevice);
    CUDA_CHECK(err);
    err = hipMemcpy(d_B, B, sizeof(float) *N2*N3, hipMemcpyHostToDevice);
    CUDA_CHECK(err);

    dim3 block(TILE_WIDTH / 2, TILE_WIDTH, 1);
    dim3 grid(ceil((float) N3/(TILE_WIDTH * 2)), ceil((float) N1/TILE_WIDTH), 1);

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    matrixMulKernel<<<grid, block>>>(d_A, d_B, d_C, N1, N2, N3);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel execution time: %.3f milliseconds\n", elapsedTime);
    
    err = hipMemcpy(C, d_C, sizeof(float) *N1*N3, hipMemcpyDeviceToHost);
    CUDA_CHECK(err);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);
    return 0;
}

__global__ void matrixMulKernel(float *A, float *B, float *C, int N1, int N2, int N3)
{
    assert(TILE_WIDTH == blockDim.x *2 );
    assert(TILE_WIDTH == blockDim.y);

    int by = blockIdx.y;
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int i = by * TILE_WIDTH + ty;
    int j = bx * TILE_WIDTH + tx * 2;

    __shared__ float sh_A[TILE_WIDTH][TILE_WIDTH + 1];
    __shared__ float sh_B[TILE_WIDTH][TILE_WIDTH + 1];

    float value = 0.0f;
    float value1 = 0.0f;

    for (int phase = 0; phase < ceil((float) N2/TILE_WIDTH);phase++)
    {
        int a_col = phase * TILE_WIDTH + tx;
        int b_row = phase * TILE_WIDTH + ty;

        if (i < N1 && a_col < N2)
        {
            sh_A[ty][tx] = A[(i*N2 + a_col)];
        }
        else 
        {
            sh_A[ty][tx] = 0.0f;
        }

        if (b_row < N2 && j < N3)
        {
            sh_B[ty][tx * 2] = B[(b_row*N3 + j)];
        }
        else
        {
            sh_B[ty][tx * 2] = 0.0f;
        }

        if (b_row < N2 && j + 1 < N3)
        {
            sh_B[ty][tx * 2 + 1] = B[b_row * N3 + j + 1];
        }
        else
        {
            sh_B[ty][tx * 2 + 1] = 0.0f;
        }
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH;k++)
        {
            value += sh_A[ty][k] * sh_B[k][tx * 2];
            value1 += sh_A[ty][k] * sh_B[k][tx * 2 + 1];
        }
        
        __syncthreads();
    }

    if (i < N1 && j < N3)
    {
        C[i*N3+j] = value;
    }
    if (i < N1 && j + 1 < N3)
    {
        C[i*N3+j+1] = value1;
    }
}
